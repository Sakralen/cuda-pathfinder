#include "hip/hip_runtime.h"
#include "pathfinder.cuh"

__forceinline__ __device__ int isVerticalAdjacentValid(int index, int offset, int *dStates, int fieldSize)
{
    return (((index + offset) >= 0) && ((index + offset) < fieldSize * fieldSize)
            // && (dStates[index + offset] != ON_FRONTIER)
            // && (dStates[index + offset] != VISITED));
            && (dStates[index + offset] == NOT_VISITED));
}

__forceinline__ __device__ int isHorizontalAdjacentValid(int index, int offset, int *dStates, int fieldSize)
{
    return ((index / fieldSize) == ((index + offset) / fieldSize)
            // && (dStates[index + offset] != ON_FRONTIER)
            // && (dStates[index + offset] != VISITED));
            && (dStates[index + offset] == NOT_VISITED));
}

__global__ void propagateWave(int dstLinearIndex, int fieldSize, int *dField, int *dStates, int *dCanPropagateFurther, int *dIsDstReached)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int linearIndex = x + y * gridDim.x * blockDim.x;

    if (linearIndex < 0 || linearIndex >= fieldSize * fieldSize)
    {
        return;
    }

    if (dStates[linearIndex] != ON_FRONTIER)
    {
        return;
    }

    //__shared__ int isBlockNotTrapped;
    int isThreadNotTrapped = FALSE;

    dStates[linearIndex] = VISITED;

    __syncthreads();

    if (isHorizontalAdjacentValid(linearIndex, -1, dStates, fieldSize))
    {
        dStates[linearIndex - 1] = ON_FRONTIER;
        dField[linearIndex - 1] = dField[linearIndex] + 1;
        isThreadNotTrapped = TRUE;
    }
    if (isHorizontalAdjacentValid(linearIndex, 1, dStates, fieldSize))
    {
        dStates[linearIndex + 1] = ON_FRONTIER;
        dField[linearIndex + 1] = dField[linearIndex] + 1;
        isThreadNotTrapped = TRUE;
    }
    if (isVerticalAdjacentValid(linearIndex, -fieldSize, dStates, fieldSize))
    {
        dStates[linearIndex - fieldSize] = ON_FRONTIER;
        dField[linearIndex - fieldSize] = dField[linearIndex] + 1;
        isThreadNotTrapped = TRUE;
    }
    if (isVerticalAdjacentValid(linearIndex, fieldSize, dStates, fieldSize))
    {
        dStates[linearIndex + fieldSize] = ON_FRONTIER;
        dField[linearIndex + fieldSize] = dField[linearIndex] + 1;
        isThreadNotTrapped = TRUE;
    }

    int didThreadReachDst = (linearIndex == dstLinearIndex);

    __syncthreads();
    atomicOr(dCanPropagateFurther, isThreadNotTrapped);

    __syncthreads();
    atomicOr(dIsDstReached, didThreadReachDst);
}

int execPathfinder(int srcLinearIndex, int dstLinearIndex, int fieldSize, int *dField, int *dStates, dim3 gridDim, dim3 blockDim, float *elapsedTime)
{
    // Setting src:
    setSingleElementOnDevice(dStates, srcLinearIndex, ON_FRONTIER);
    setSingleElementOnDevice(dField, srcLinearIndex, 0);

    // Setting dst:
    setSingleElementOnDevice(dStates, dstLinearIndex, NOT_VISITED);

    // Setting flags:
    int *dCanPropagateFurther = NULL;
    hipMalloc(&dCanPropagateFurther, sizeof(int));
    // hipMemset(dCanPropagateFurther, FALSE, sizeof(int));

    int hCanPropagateFurther = FALSE;

    int *dIsDstReached = NULL;
    hipMalloc(&dIsDstReached, sizeof(int));
    // hipMemset(dIsDstReached, FALSE, sizeof(int));

    int hIsDstReached = FALSE;

    hipEvent_t start, stop;
    //float elapsedTime;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);

    // Pathfinder:
    do
    {
        hipMemset(dCanPropagateFurther, FALSE, sizeof(int));
        hipMemset(dIsDstReached, FALSE, sizeof(int));

        propagateWave<<<gridDim, blockDim>>>(dstLinearIndex, fieldSize, dField, dStates, dCanPropagateFurther, dIsDstReached);

        hipMemcpy(&hCanPropagateFurther, dCanPropagateFurther, sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(&hIsDstReached, dIsDstReached, sizeof(int), hipMemcpyDeviceToHost);

#ifdef DEBUG
        int fieldBytes = fieldSize * fieldSize * sizeof(int);

        int *hField = (int *)malloc(fieldBytes);
        hipMemcpy(hField, dField, fieldBytes, hipMemcpyDeviceToHost);
        printField(hField, fieldSize);
        printf("\n");
        hipMemcpy(hField, dStates, fieldBytes, hipMemcpyDeviceToHost);
        printField(hField, fieldSize);
        free(hField);

        printf("hCanPropagateFurther val: %d\n", hCanPropagateFurther);
        printf("hIsDstReached val: %d\n", hIsDstReached);
        printf("\n");
#endif
    } while ((hIsDstReached == FALSE) && (hCanPropagateFurther == TRUE)); // TODO: Probably should check this condition

    hipEventRecord(stop, 0);

    hipEventSynchronize(stop);
    hipEventElapsedTime(elapsedTime, start, stop);

    int pathLength = 0;
    hipMemcpy(&pathLength, &dField[dstLinearIndex], sizeof(int), hipMemcpyDeviceToHost);

    hipFree(dCanPropagateFurther);
    hipFree(dIsDstReached);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    return pathLength;
}

void generateSrcAndDest(int *srcLinearIndex, int *dstLinearIndex, int fieldSize)
{
    do
    {
        *srcLinearIndex = rand() % (fieldSize * fieldSize);
        *dstLinearIndex = rand() % (fieldSize * fieldSize);
    } while (abs(*srcLinearIndex - *dstLinearIndex) < DELTA_SRC_DST);
}
