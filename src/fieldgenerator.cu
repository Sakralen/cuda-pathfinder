#include "hip/hip_runtime.h"
#include "fieldgenerator.cuh"

__global__ void init(unsigned int seed, hiprandState_t* curandStates, int fieldSize) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int offset = x + y * gridDim.x * blockDim.x;

    if (offset < 0 || offset >= fieldSize * fieldSize) {
        return;
    }

    hiprand_init(seed, offset, 0, &curandStates[offset]);
}

__global__ void generate(hiprandState_t* curandStates, int* dField, int* dStates, int fieldSize) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int offset = x + y * gridDim.x * blockDim.x;

    if (offset < 0 || offset >= fieldSize * fieldSize) {
        return;
    }

    int generated = (hiprand(&curandStates[offset]) % (RAND_HIGH - RAND_LOW + 1)) + RAND_LOW;
    dField[offset] = (generated == -1) ? BARRIER : 0; 
    dStates[offset] = (generated == -1) ? BARRIER : NOT_VISITED;
}

// hiprandState_t* curandStates has nothing to do with dStates.
// This pointer is necessary part of cuRAND execution.
int generateField(int* dField, int* dStates, int fieldSize, dim3* gridDimStruct, dim3* blockDimStruct) {
    // int fieldBytes = fieldSize * fieldSize * sizeof(int);
    hiprandState_t* curandStates; 

    if (hipMalloc(&curandStates, fieldSize * fieldSize * sizeof(hiprandState_t)) != hipSuccess) {
        printf("Error: failed to allocate curandStates[] on the device\n");
        return FALSE;
    }

    init<<<*gridDimStruct, *blockDimStruct>>>(time(NULL), curandStates, fieldSize);
    generate<<<*gridDimStruct, *blockDimStruct>>>(curandStates, dField, dStates, fieldSize);

    // if (hipMemcpy(dStates, dField, fieldBytes, hipMemcpyDeviceToDevice) != hipSuccess) {
    //     printf("Error: failed to copy dField[] to dStates[]\n");

    //     hipFree(curandStates);
    //     return FALSE;
    // }

    hipFree(curandStates);
    return TRUE;
}
