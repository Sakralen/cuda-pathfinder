#include <stdio.h>
#include <stdlib.h>

#include <hip/hip_runtime.h>


__global__ void kernel(int* dField, int fieldSize) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int linearIndex = x + y * gridDim.x * blockDim.x;

    while (!(linearIndex < 0 || linearIndex >= fieldSize * fieldSize)) {
        dField[linearIndex] = 1;
        linearIndex += gridDim.x;
    }
}

void printField(int *field, int size)
{
    for (int i = 0; i < size; i++)
    {
        for (int j = 0; j < size; j++)
        {
            printf("%2d ", field[i * size + j]);
        }
        printf("\n");
    }
}

int main(int argc, char **argv) {

    int fieldSize = atoi(argv[1]);
    int gridDimVal = atoi(argv[2]);
    int blockDimVal = atoi(argv[3]);

    int fieldBytes = fieldSize * fieldSize * sizeof(int);

    int* hField = (int*)malloc(fieldBytes);

    int* dField;
    hipMalloc(&dField, fieldBytes);
    hipMemset(dField, 0, fieldBytes);

    hipMemcpy(hField, dField, fieldBytes, hipMemcpyDeviceToHost);
    // printField(hField, fieldSize);
    // printf("\n");
    
    dim3 gridDim(gridDimVal, gridDimVal);
    dim3 blockDim(blockDimVal, blockDimVal);

    hipEvent_t start, stop;
    float elapsedTime;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);

    kernel<<<gridDim, blockDim>>>(dField, fieldSize);

    hipEventRecord(stop, 0);

    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);

    hipMemcpy(hField, dField, fieldBytes, hipMemcpyDeviceToHost);
    // printField(hField, fieldSize);
    printf("\n");
    printf("Elapsed time is %.5f\n", elapsedTime);

    hipFree(dField);
    free(hField);

    return 0;
}
